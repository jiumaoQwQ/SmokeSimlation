#include "hip/hip_runtime.h"
#include "SmokeSim.h"
#include "CudaArray.cuh"
#include "hip/hip_vector_types.h"

#ifndef N
#define N 128
#endif

template <class T>
__global__ void fill_zero_kernel(CudaSurfaceAccessor<T> suracc)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;

    suracc.write(T{0}, x, y, z);
}

__global__ void change_velocity_density_kernel(CudaSurfaceAccessor<float> density, CudaSurfaceAccessor<float4> velocity, CudaSurfaceAccessor<float> bound)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;
    if (bound.read(x, y, z) > 0)
    {
        density.write(1, x, y, z);
        velocity.write(make_float4(0, 0, 0.5f, 0), x, y, z);
    }
}

__global__ void advect_kernel(CudaSurfaceAccessor<float4> nextPos, CudaTextureAccessor<float4> velocity, CudaTextureAccessor<float> bound)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;

    auto sample = [](CudaTextureAccessor<float4> velAcc, float3 pos, float3 vel, float time) -> float3
    {
        pos = pos - vel * time;
        float4 res = velAcc.sample(pos.x, pos.y, pos.z);
        return make_float3(res.x, res.y, res.z);
    };

    float3 pos = make_float3(x + 0.5f, y + 0.5f, z + 0.5f);

    if (bound.sample(x, y, z) < 0)
    {
        float3 k1 = sample(velocity, pos, make_float3(0, 0, 0), 0);
        float3 k2 = sample(velocity, pos, k1, 0.5f);
        float3 k3 = sample(velocity, pos, k2, 0.5f);
        float3 k4 = sample(velocity, pos, k3, 1);

        pos -= (k1 + 2 * k2 + 2 * k3 + k4) / 6.0f;
    }
    nextPos.write(make_float4(pos.x, pos.y, pos.z, 0), x, y, z);
}

template <class T>
__global__ void sample_kernel(CudaSurfaceAccessor<T> sufAcc, CudaTextureAccessor<T> texAcc, CudaSurfaceAccessor<float4> nextPos)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;

    float4 next_pos = nextPos.read(x, y, z);
    T res = texAcc.sample(next_pos.x, next_pos.y, next_pos.z);
    sufAcc.write(res, x, y, z);
}

__global__ void cal_div_v(CudaSurfaceAccessor<float> div_v, CudaSurfaceAccessor<float4> v)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;

    float vr = v.read<hipBoundaryModeClamp>(x + 1, y, z).x;
    float vl = v.read<hipBoundaryModeClamp>(x - 1, y, z).x;
    float vu = v.read<hipBoundaryModeClamp>(x, y + 1, z).y;
    float vd = v.read<hipBoundaryModeClamp>(x, y - 1, z).y;
    float vf = v.read<hipBoundaryModeClamp>(x, y, z + 1).z;
    float vb = v.read<hipBoundaryModeClamp>(x, y, z - 1).z;
    div_v.write((vr - vl + vu - vd + vf - vb) * 0.5f, x, y, z);
}

__global__ void jacobi_kernel(CudaSurfaceAccessor<float> pressureNext, CudaSurfaceAccessor<float> pressure, CudaSurfaceAccessor<float> div_v)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;
    float r = pressure.read<hipBoundaryModeClamp>(x + 1, y, z);
    float l = pressure.read<hipBoundaryModeClamp>(x - 1, y, z);
    float u = pressure.read<hipBoundaryModeClamp>(x, y + 1, z);
    float d = pressure.read<hipBoundaryModeClamp>(x, y - 1, z);
    float f = pressure.read<hipBoundaryModeClamp>(x, y, z + 1);
    float b = pressure.read<hipBoundaryModeClamp>(x, y, z - 1);

    float div = div_v.read<hipBoundaryModeClamp>(x, y, z);
    float res = (r + l + u + d + f + b - div) / 6.0f;
    pressureNext.write(res, x, y, z);
}

__global__ void projection_kernel(CudaSurfaceAccessor<float4> v, CudaSurfaceAccessor<float> pressure)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= 128 || y >= 128 || z >= 128)
        return;

    float r = pressure.read<hipBoundaryModeClamp>(x + 1, y, z);
    float l = pressure.read<hipBoundaryModeClamp>(x - 1, y, z);
    float u = pressure.read<hipBoundaryModeClamp>(x, y + 1, z);
    float d = pressure.read<hipBoundaryModeClamp>(x, y - 1, z);
    float f = pressure.read<hipBoundaryModeClamp>(x, y, z + 1);
    float b = pressure.read<hipBoundaryModeClamp>(x, y, z - 1);

    float4 vel = v.read(x, y, z);
    vel.x -= (r - l) * 0.5f;
    vel.y -= (u - d) * 0.5f;
    vel.z -= (f - b) * 0.5f;
    v.write(vel, x, y, z);
}

struct SmokeSim::SmokeSimImpl
{
    std::unique_ptr<CudaTexture<float>> density;
    std::unique_ptr<CudaTexture<float4>> velocity;
    std::unique_ptr<CudaTexture<float>> bound;
    std::unique_ptr<CudaSurface<float4>> nextPos;
    std::unique_ptr<CudaSurface<float>> div_v;
    std::unique_ptr<CudaTexture<float>> pressure;

    std::unique_ptr<CudaTexture<float>> float_buff;
    std::unique_ptr<CudaTexture<float4>> float4_buff;

    uint3 dim;
};

SmokeSim::SmokeSim(unsigned int x, unsigned int y, unsigned int z)
{
    Impl = std::make_unique<SmokeSimImpl>();
    Impl->density = std::make_unique<CudaTexture<float>>(uint3{x, y, z});
    Impl->velocity = std::make_unique<CudaTexture<float4>>(uint3{x, y, z});
    Impl->bound = std::make_unique<CudaTexture<float>>(uint3{x, y, z});
    Impl->nextPos = std::make_unique<CudaSurface<float4>>(uint3{x, y, z});
    Impl->div_v = std::make_unique<CudaSurface<float>>(uint3{x, y, z});
    Impl->pressure = std::make_unique<CudaTexture<float>>(uint3{x, y, z});

    Impl->float_buff = std::make_unique<CudaTexture<float>>(uint3{x, y, z});
    Impl->float4_buff = std::make_unique<CudaTexture<float4>>(uint3{x, y, z});

    Impl->dim = uint3{x, y, z};

    fill_zero_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(Impl->pressure->accessSurface());
}

SmokeSim::~SmokeSim()
{
}

void SmokeSim::copyInDensity(float *data)
{
    Impl->density->copyIn(data);
}

void SmokeSim::copyInVelocity(float *data)
{
    Impl->velocity->copyIn((float4 *)data);
}

void SmokeSim::copyInBound(float *data)
{
    Impl->bound->copyIn(data);
}

void SmokeSim::copyOutDensity(float *data)
{
    Impl->density->copyOut(data);
}

void SmokeSim::advect()
{
    fill_zero_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->pressure->accessSurface());

    change_velocity_density_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->density->accessSurface(),Impl->velocity->accessSurface(),Impl->bound->accessSurface());

    advect_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->nextPos->accessSurface(), Impl->velocity->accessTexture(), Impl->bound->accessTexture());

    // sample for density
    sample_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->float_buff->accessSurface(), Impl->density->accessTexture(), Impl->nextPos->accessSurface());

    std::swap(Impl->float_buff, Impl->density);

    // sample for velocity
    sample_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->float4_buff->accessSurface(), Impl->velocity->accessTexture(), Impl->nextPos->accessSurface());

    std::swap(Impl->float4_buff, Impl->velocity);
}

void SmokeSim::projection()
{
    cal_div_v<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->div_v->accessSurface(), Impl->velocity->accessSurface());

    int times = 50;
    for (int i = 0; i < times; i++)
    {
        jacobi_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
            Impl->float_buff->accessSurface(), Impl->pressure->accessSurface(),
            Impl->div_v->accessSurface());
        std::swap(Impl->float_buff, Impl->pressure);
    }

    projection_kernel<<<{(N + 7) / 8, (N + 7) / 8, (N + 7) / 8}, {8, 8, 8}>>>(
        Impl->velocity->accessSurface(), Impl->pressure->accessSurface());
}

void SmokeSim::step()
{
    for (int i = 0; i < 24; i++)
    {
        advect();
        projection();
    }
}
